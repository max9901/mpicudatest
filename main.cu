#include <iostream>
#include <mpi.h>
#include <mpi-ext.h>

#define MPI_CHECK_RETURN(error_code) {                                           \
    if (error_code != MPI_SUCCESS) {                                             \
        char error_string[BUFSIZ];                                               \
        int length_of_error_string;                                              \
        int world_rank;                                                          \
        MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);                              \
        MPI_Error_string(error_code, error_string, &length_of_error_string);     \
        fprintf(stderr, "%3d: %s\n", world_rank, error_string);                  \
        exit(1);                                                                 \
    }}

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
        int world_rank;                                                          \
        MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);                              \
        char * name = (char*) malloc (MPI_MAX_PROCESSOR_NAME * sizeof(char));     \
        int name_len;                                                            \
        MPI_Get_processor_name(name, &name_len);                                 \
		fprintf(stderr, "%3d %s: CUDA Error %s at line %d in file %s\n",	\
             world_rank,name,hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("%3d %s: CUDA Error %s at line %d in file %s\n",		            \
			 world_rank,name,hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }

int main() {
        int test = 0;
        int N = 100;

        printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
        printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
        printf("This MPI library does not have CUDA-aware support.\n");
#else
        printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

        printf("Run time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
        if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
        printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */


        MPI_CHECK_RETURN(MPI_Init_thread(NULL, NULL,MPI_THREAD_FUNNELED, &test));
        if(test != MPI_THREAD_FUNNELED){
            std::cout << "Somethings is wrong with the mpi init " << test << " != MPI_THREAD_FUNNELED (1)\n";
        }
        int world_size;
        int world_rank;
        MPI_CHECK_RETURN(MPI_Comm_size(MPI_COMM_WORLD, &world_size));
        MPI_CHECK_RETURN(MPI_Comm_rank(MPI_COMM_WORLD, &world_rank));

        int *a,*check;
        int *d_a;

        // Allocate host memory
        a   = (int*)malloc(sizeof(int) * N);
        check   = (int*)malloc(sizeof(int) * N);

        int offset = N/world_size*world_rank;
        int sizedit = N/world_size;
        // Initialize host arrays

        for(int i = 0; i < N ; i++) {
            a[i] = 0;
            check[i] = i;
        }
        for(int i = offset; i < N/world_size + offset; i++){
            a[i] = i;
        }

        // Allocate device memory
        CUDA_CHECK_RETURN(hipMalloc((void**)&d_a, sizeof(float) * N));
        
        // Transfer data from host to device memory
        CUDA_CHECK_RETURN(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));


        CUDA_CHECK_RETURN(hipDeviceSynchronize());

        MPI_CHECK_RETURN(MPI_Barrier(MPI_COMM_WORLD));
        std::cout << world_rank << " :check the cpu allgather " << a << "\n";
        MPI_CHECK_RETURN(MPI_Allgather(
                &a[offset],                            //sendbuffer
                sizedit,                               //sendcount
                MPI_INT,                               //type
                a,                                     //receivebuffer
                sizedit,                               //recvcount (from any process)
                MPI_INT,                               //type
                MPI_COMM_WORLD));                      //handle


        for(int i = 0; i < N ; i++) {
            if(a[i] != check[i]){
                printf("ERROR");
                exit(0);
            }
        }

        MPI_CHECK_RETURN(MPI_Barrier(MPI_COMM_WORLD));
        std::cout << "\n";
        MPI_CHECK_RETURN(MPI_Barrier(MPI_COMM_WORLD));

        std::cout << world_rank << " : check the GPU allgather " << d_a << "\t"  << "\n";
        MPI_CHECK_RETURN(MPI_Allgather(
                &d_a[offset],                           //sendbuffer
                sizedit,                              //sendcount
                MPI_INT,                              //type
                d_a,                                  //receivebuffer
                sizedit,                              //recvcount (from any process)
                MPI_INT,                              //type
                MPI_COMM_WORLD));                     //handle

        CUDA_CHECK_RETURN(hipMemcpy(a, d_a, sizeof(float) * N, hipMemcpyDeviceToHost));
        
        CUDA_CHECK_RETURN(hipDeviceSynchronize());

        for(int i = 0; i < N ; i++) {
            if(a[i] != check[i]){
                printf("ERROR");
                exit(0);
            }
        }

        MPI_Finalize();

        exit(0);
        return true;
}
